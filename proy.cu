#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define pi 3.14159265359

void load_image(char *fname, int Nx, int Ny, float *img)
{
  FILE *fp;

  fp = fopen(fname, "r");

  for (int i = 0; i < Ny; i++)
  {
    for (int j = 0; j < Nx; j++)
      fscanf(fp, "%f ", &img[i * Nx + j]);
    fscanf(fp, "\n");
  }

  fclose(fp);
}

void save_image(char *fname, int Nx, int Ny, float *img)
{
  FILE *fp;

  fp = fopen(fname, "w");
  
  for (int i = 0; i < Ny; i++)
  {
    for (int j = 0; j < Nx; j++){
      fprintf(fp, "%10.3f ", img[i * Nx + j]);
    
    }
    fprintf(fp, "\n");
  }

  fclose(fp);
}



void calculate_kernel(int kernel_size, float sigma, float *kernel)
{

  int Nk2 = kernel_size * kernel_size;
  float x, y, center;

  center = (kernel_size - 1) / 2.0;

  for (int i = 0; i < Nk2; i++)
  {
    x = (float)(i % kernel_size) - center;
    y = (float)(i / kernel_size) - center;
    kernel[i] = -(1.0 / pi * pow(sigma, 4)) * (1.0 - 0.5 * (x * x + y * y) / (sigma * sigma)) * exp(-0.5 * (x * x + y * y) / (sigma * sigma));
  }
}


void conv_img_cpu(float *img, float *kernel, float *imgf, int Nx, int Ny, int kernel_size)
{

  float sum = 0;
  int center = (kernel_size - 1) / 2;
  ;
  int ii, jj;

  for (int i = center; i < (Ny - center); i++)
    for (int j = center; j < (Nx - center); j++)
    {
      sum = 0;
      for (int ki = 0; ki < kernel_size; ki++)
        for (int kj = 0; kj < kernel_size; kj++)
        {
          ii = kj + j - center;
          jj = ki + i - center;
          sum += img[jj * Nx + ii] * kernel[ki * kernel_size + kj];
        }
      

      imgf[i * Nx + j] = sum;
    }
}


__global__ void conv_img_gpu(float *img, float *kernel, float *imgf, int Nx, int Ny, int kernel_size)
{

  int tid = threadIdx.x;
  int iy = blockIdx.x + (kernel_size - 1) / 2;

  int idx = iy * Nx;
  int K2 = kernel_size * kernel_size;
  int center = (kernel_size - 1) / 2;
  int ii, jj;
  float sum = 0.0;
  extern __shared__ float sdata[];

  if (idx < Ny*Nx )
  {
    for(int i = center;i<(Nx - center);i++){
      sum = 0;
      for (int ki = 0; ki < kernel_size; ki++)
        for (int kj = 0; kj < kernel_size; kj++)
        {
          ii = kj + i - center;
          jj = ki + iy - center;
          if(jj * Nx + ii < Nx*Ny)
          sum += img[jj * Nx + ii] * kernel[ki * kernel_size + kj];
        }

      imgf[idx+i] = sum;
   }
  }
}




#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}







int main(int argc, char *argv[])
{
  hipEvent_t start, stop;
  clock_t t1, t2;
  double ms;


  float milliseconds = 0;
  int Nx, Ny;
  int kernel_size;
  float sigma;
  char finput[256], foutput[256], foutput_cpu[256];
  int Nblocks, Nthreads;

  sprintf(finput, "dog.dat");

  sprintf(foutput, "gpu_output.dat");
  sprintf(foutput_cpu, "cpu_output.dat");


  Nx = 750;
  Ny = 750;

  kernel_size = 3;
  sigma = 0.55;
  


  float *img, *imgf, *imgf_cpu, *kernel;

  img = (float *)malloc(Nx * Ny * sizeof(float));
  imgf = (float *)malloc(Nx * Ny * sizeof(float));
  imgf_cpu = (float *)malloc(Nx * Ny * sizeof(float));
  kernel = (float *)malloc(kernel_size * kernel_size * sizeof(float));

  //Cargamos la imagen y calculamos los valores del kennel segun el kernel_size
  load_image(finput, Nx, Ny, img);
  calculate_kernel(kernel_size, sigma, kernel);

  t1 = clock();
 



  conv_img_cpu(img, kernel, imgf_cpu, Nx, Ny, kernel_size);

  t2 = clock();
  ms = 1000.0 * (double)(t2 - t1) / CLOCKS_PER_SEC;
  printf("Tiempo (CPU): %f[ms]\n", ms);



 



  float *d_img, *d_imgf, *d_kernel;

  hipMalloc(&d_img, Nx * Ny * sizeof(float));
  hipMalloc(&d_imgf, Nx * Ny * sizeof(float));
  hipMalloc(&d_kernel, kernel_size * kernel_size * sizeof(float));




  hipMemcpy(d_img, img, Nx * Ny * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, kernel, kernel_size * kernel_size * sizeof(float), hipMemcpyHostToDevice);

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  
  int block_size = 256;


  
  int grid_size = (Ny - (kernel_size - 1));


  conv_img_gpu<<<grid_size, 1, kernel_size * kernel_size * sizeof(float)>>>(d_img, d_kernel, d_imgf, Nx, Ny, kernel_size);    


  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  
  printf("Tiempo (GPU): %f ms\n", milliseconds);

  gpuErrchk(hipMemcpy(imgf, d_imgf, Nx * Ny * sizeof(float), hipMemcpyDeviceToHost));




  save_image(foutput, Nx, Ny, imgf);
  save_image(foutput_cpu, Nx, Ny, imgf_cpu);



  free(img);
  free(imgf);
  free(kernel);

  hipFree(d_img);
  hipFree(d_imgf);
  hipFree(d_kernel);
}
