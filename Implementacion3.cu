#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define pi 3.14159265359

void load_image(char *fname, int Nx, int Ny, float *img)
{
  FILE *fp;

  fp = fopen(fname, "r");

  for (int i = 0; i < Ny; i++)
  {
    for (int j = 0; j < Nx; j++)
      fscanf(fp, "%f ", &img[i * Nx + j]);
    fscanf(fp, "\n");
  }

  fclose(fp);
}

void save_image(char *fname, int Nx, int Ny, float *img)
{
  FILE *fp;

  fp = fopen(fname, "w");

  for (int i = 0; i < Ny; i++)
  {
    for (int j = 0; j < Nx; j++)
      fprintf(fp, "%10.3f ", img[i * Nx + j]);
    fprintf(fp, "\n");
  }

  fclose(fp);
}


void calculate_kernel(int kernel_size, float sigma, float *kernel)
{

  int Nk2 = kernel_size * kernel_size;
  float x, y, center;

  center = (kernel_size - 1) / 2.0;

  for (int i = 0; i < Nk2; i++)
  {
    x = (float)(i % kernel_size) - center;
    y = (float)(i / kernel_size) - center;
    kernel[i] = -(1.0 / pi * pow(sigma, 4)) * (1.0 - 0.5 * (x * x + y * y) / (sigma * sigma)) * exp(-0.5 * (x * x + y * y) / (sigma * sigma));
  }
}


void get_all_matrices(float *img, float *out_img, int m, int n, int kernel_size)
{
  for(int i = 0; i < (m - kernel_size + 1); i++){
    for(int j = 0; j < (n - kernel_size + 1); j++){
      for (int k = 0; k < kernel_size; k++){
        out_img[k] = img[k];
        out_img[k + kernel_size*1] = img[k + kernel_size*1];
        out_img[k + kernel_size*2] = img[k + kernel_size*2];
      }
    }
  }
}


void conv_img_cpu(float *img, float *kernel, float *imgf, int Nx, int Ny, int kernel_size)
{

  float sum = 0;
  int center = (kernel_size - 1) / 2;
  ;
  int ii, jj;

  for (int i = center; i < (Ny - center); i++)
    for (int j = center; j < (Nx - center); j++)
    {
      sum = 0;
      for (int ki = 0; ki < kernel_size; ki++)
        for (int kj = 0; kj < kernel_size; kj++)
        {
          ii = kj + j - center;
          jj = ki + i - center;
          sum += img[jj * Nx + ii] * kernel[ki * kernel_size + kj];
        }
      

      imgf[i * Nx + j] = sum;
    }
}


// __global__ void conv_img_gpu(float *img, float *kernel, float *imgf, int Nx, int Ny, int kernel_size,int offset)
// {
 
//   int tid = threadIdx.x;
 

//   int iy = blockIdx.x + (kernel_size - 1) / 2;

//   int ix = threadIdx.x + (kernel_size - 1) / 2+offset;

//   int idx = iy * Nx + ix;

//  int K2 = kernel_size * kernel_size;

//   int center = (kernel_size - 1) / 2;


//   int ii, jj;
//   float sum = 0.0;


//   extern __shared__ float sdata[];

 

//   if (tid < K2)
//     sdata[tid] = kernel[tid];


//   __syncthreads();


//   if(ix > Nx){
//     return;    
//   }

//   if (idx < Nx * Ny)
//   {
//     for (int ki = 0; ki < kernel_size; ki++)
//       for (int kj = 0; kj < kernel_size; kj++)
//       {
//         ii = kj + ix - center;
//         jj = ki + iy - center;
//         sum += img[jj * Nx + ii] * sdata[ki * kernel_size + kj];
//       }
    
//     imgf[idx] = sum;
//   }
// }


// __global__ void conv_img_gpu_f2(float *img, float *kernel, float *imgf, int Nx, int Ny, int kernel_size,int offset)
// {
 
//   int tid = threadIdx.x;
 

//   int iy = blockIdx.x + (kernel_size - 1) / 2;

//   int ix = threadIdx.x + (kernel_size - 1) / 2+offset;

//   int idx = iy * Nx + ix;

 
//   int K2 = kernel_size * kernel_size;

//   int center = (kernel_size - 1) / 2;


//   int ii, jj;
//   float sum = 0.0;

//   if(ix > Nx){
//     return;    
//   }

//   if (idx < Nx * Ny)
//   {
//     for (int ki = 0; ki < kernel_size; ki++)
//       for (int kj = 0; kj < kernel_size; kj++)
//       {
//         ii = kj + ix - center;
//         jj = ki + iy - center;
//         sum += img[jj * Nx + ii] * kernel[ki * kernel_size + kj];
//       }
    
//     imgf[idx] = sum;
//   }
// }


__global__ void conv_img_gpu_f3(float *img, float *kernel, float *imgf, int Nx, int Ny, int kernel_size)
{
 
  int tid = threadIdx.x;
  int final_size = (Nx - kernel_size + 1) * (Ny - kernel_size + 1);
  float sum = 0.0;  

  extern __shared__ float simg[];
  
  if (tid < final_size) {
    for (int i = 0; i < kernel_size * kernel_size; i++)
      simg[tid * (kernel_size * kernel_size) + i] = img[tid * (kernel_size * kernel_size) + i];
  }

  __syncthreads();

  if (tid < final_size) {
    for (int i = 0; i < kernel_size * kernel_size; i++){
        sum += simg[tid * (kernel_size * kernel_size) + i] * kernel[i];
    }
   
    imgf[tid] = sum;
  }
  
}

int main(int argc, char *argv[])
{
  hipEvent_t start, stop;
  clock_t t1, t2;
  double ms;


  float milliseconds = 0;
  int Nx, Ny;
  int kernel_size;
  float sigma;
  char finput[256], foutput[256], foutput_gpu_3[256],foutput_cpu[256];
  int Nblocks, Nthreads;

  sprintf(finput, "dog.dat");

  sprintf(foutput_gpu_3, "gpu_3_output.dat");
 
  sprintf(foutput_cpu, "cpu_output.dat");


  Nx = 750;
  Ny = 750;

  kernel_size = 3;
  sigma = 0.55;
  


  float *img, *imgf, *imgf_cpu, *kernel;

  img = (float *)malloc(Nx * Ny * sizeof(float));
  imgf = (float *)malloc(Nx * Ny * sizeof(float));
  imgf_cpu = (float *)malloc(Nx * Ny * sizeof(float));


  kernel = (float *)malloc(kernel_size * kernel_size * sizeof(float));

  //Cargamos la imagen y calculamos los valores del kernel segun el kernel_size
  load_image(finput, Nx, Ny, img);
  calculate_kernel(kernel_size, sigma, kernel);

  t1 = clock();
 



  conv_img_cpu(img, kernel, imgf_cpu, Nx, Ny, kernel_size);

  t2 = clock();
  ms = 1000.0 * (double)(t2 - t1) / CLOCKS_PER_SEC;
  printf("Tiempo (CPU): %f[ms]\n", ms);


  float *d_img, *d_imgf, *d_kernel;

  hipMalloc(&d_img, Nx * Ny * sizeof(float));
  hipMalloc(&d_imgf, Nx * Ny * sizeof(float));
  hipMalloc(&d_kernel, kernel_size * kernel_size * sizeof(float));




  hipMemcpy(d_img, img, Nx * Ny * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, kernel, kernel_size * kernel_size * sizeof(float), hipMemcpyHostToDevice);

  Nblocks = Ny - (kernel_size - 1);
  Nthreads = Nx - (kernel_size - 1);



  //redundant - coalescent version
  float *d_r_img;

  float *r_img = (float *)malloc(kernel_size * kernel_size * Nx * Ny * sizeof(float));
  get_all_matrices(img, r_img, Ny, Nx, kernel_size);
  
 


  hipMalloc(&d_r_img, kernel_size * kernel_size * Nx * Ny * sizeof(float));
  hipMemcpy(d_r_img, r_img, kernel_size * kernel_size * Nx * Ny * sizeof(float), hipMemcpyHostToDevice);
  int block_size=256;
  int n_elems = (Nx - kernel_size + 1) * (Ny - kernel_size + 1);
  int grid_size = (int)ceil((float)n_elems / block_size);

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  
  conv_img_gpu_f3<<<grid_size, block_size, kernel_size * kernel_size * Nx * Ny * sizeof(float)>>>(d_r_img, d_kernel, d_imgf, Nx, Ny, kernel_size);    
 
 
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("Tiempo Implementacion 3 (GPU): %f ms\n", milliseconds);

  //

  float *imgf_gpu_3;
  imgf_gpu_3 = (float *)malloc(Nx * Ny * sizeof(float));
  hipMemcpy(imgf_gpu_3, d_imgf, Nx * Ny * sizeof(float), hipMemcpyDeviceToHost);


 
  save_image(foutput_gpu_3, Nx, Ny, imgf_gpu_3);
  save_image(foutput_cpu, Nx, Ny, imgf_cpu);


  free(img);
  free(imgf);
  free(kernel);
  free(r_img);

  hipFree(d_img);
  hipFree(d_imgf);
  hipFree(d_kernel);
}
